#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
#include <algo/autolykos/autolykos.hpp>
#include <common/cast.hpp>
#include <benchmark/result.hpp>

///////////////////////////////////////////////////////////////////////////////
#include <benchmark/cuda/common/common.cuh>
#include <benchmark/cuda/blake2b/mhssamadani_blake2b.cuh>


__constant__ uint32_t bound_[8];


__constant__ uint64_t ivals[8] =
{
    0x6A09E667F2BDC928,
    0xBB67AE8584CAA73B,
    0x3C6EF372FE94F82B,
    0xA54FF53A5F1D36F1,
    0x510E527FADE682D1,
    0x9B05688C2B3E6C1F,
    0x1F83D9ABFB41BD6B,
    0x5BE0CD19137E2179
};


__device__ __forceinline__
uint64_t devROTR64(uint64_t b, int offset)
{
    uint2 a;
    uint2 result;
    a = vectorize(b);

    if (offset < 32)
    {
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
    }
    else
    {
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
    }
    return devectorize(result);
}


__device__ __forceinline__
void devB2B_G(
    uint64_t* v,
    int const a,
    int const b,
    int const c,
    int const d,
    uint64_t const x,
    uint64_t const y)
{
    ((uint64_t*)(v))[a] += ((uint64_t*)(v))[b] + x;
    ((uint64_t*)(v))[d] = devROTR64(((uint64_t*)(v))[d] ^ ((uint64_t*)(v))[a], 32);
    ((uint64_t*)(v))[c] += ((uint64_t*)(v))[d];
    ((uint64_t*)(v))[b] = devROTR64(((uint64_t*)(v))[b] ^ ((uint64_t*)(v))[c], 24);
    ((uint64_t*)(v))[a] += ((uint64_t*)(v))[b] + y;
    ((uint64_t*)(v))[d] = devROTR64(((uint64_t*)(v))[d] ^ ((uint64_t*)(v))[a], 16);
    ((uint64_t*)(v))[c] += ((uint64_t*)(v))[d];
    ((uint64_t*)(v))[b] = devROTR64(((uint64_t*)(v))[b] ^ ((uint64_t*)(v))[c], 63);
}


__device__ __forceinline__
void devB2B_MIX(
    uint64_t* v,
    uint64_t* m)
{
    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[0], ((uint64_t*)(m))[1]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[2], ((uint64_t*)(m))[3]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[4], ((uint64_t*)(m))[5]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[6], ((uint64_t*)(m))[7]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[8], ((uint64_t*)(m))[9]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[10], ((uint64_t*)(m))[11]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t*)(m))[12], ((uint64_t*)(m))[13]);
    devB2B_G(v, 3, 4,  9, 14, ((uint64_t*)(m))[14], ((uint64_t*)(m))[15]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t*)(m))[14], ((uint64_t*)(m))[10]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[4], ((uint64_t*)(m))[8]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[9], ((uint64_t*)(m))[15]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[13], ((uint64_t*)(m))[6]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[1], ((uint64_t*)(m))[12]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[0], ((uint64_t*)(m))[2]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[11], ((uint64_t*)(m))[7]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[5], ((uint64_t*)(m))[3]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[11], ((uint64_t*)(m))[8]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[12], ((uint64_t*)(m))[0]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[5], ((uint64_t*)(m))[2]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[15], ((uint64_t*)(m))[13]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[10], ((uint64_t*)(m))[14]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[3], ((uint64_t*)(m))[6]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[7], ((uint64_t*)(m))[1]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[9], ((uint64_t*)(m))[4]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[7], ((uint64_t*)(m))[9]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[3], ((uint64_t*)(m))[1]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[13], ((uint64_t*)(m))[12]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[11], ((uint64_t*)(m))[14]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[2], ((uint64_t*)(m))[6]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[5], ((uint64_t*)(m))[10]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[4], ((uint64_t*)(m))[0]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[15], ((uint64_t*)(m))[8]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[9], ((uint64_t*)(m))[0]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[5], ((uint64_t*)(m))[7]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[2], ((uint64_t*)(m))[4]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[10], ((uint64_t*)(m))[15]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[14], ((uint64_t*)(m))[1]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[11], ((uint64_t*)(m))[12]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[6], ((uint64_t*)(m))[8]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[3], ((uint64_t*)(m))[13]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[2], ((uint64_t*)(m))[12]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[6], ((uint64_t*)(m))[10]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[0], ((uint64_t*)(m))[11]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[8], ((uint64_t*)(m))[3]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[4], ((uint64_t*)(m))[13]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[7], ((uint64_t*)(m))[5]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t*)(m))[15], ((uint64_t*)(m))[14]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[1], ((uint64_t*)(m))[9]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[12], ((uint64_t*)(m))[5]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[1], ((uint64_t*)(m))[15]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[14], ((uint64_t*)(m))[13]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[4], ((uint64_t*)(m))[10]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[0], ((uint64_t*)(m))[7]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[6], ((uint64_t*)(m))[3]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[9], ((uint64_t*)(m))[2]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[8], ((uint64_t*)(m))[11]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t*)(m))[13], ((uint64_t*)(m))[11]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[7], ((uint64_t*)(m))[14]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[12], ((uint64_t*)(m))[1]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[3], ((uint64_t*)(m))[9]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[5], ((uint64_t*)(m))[0]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[15], ((uint64_t*)(m))[4]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[8], ((uint64_t*)(m))[6]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[2], ((uint64_t*)(m))[10]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[6], ((uint64_t*)(m))[15]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[14], ((uint64_t*)(m))[9]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[11], ((uint64_t*)(m))[3]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[0], ((uint64_t*)(m))[8]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[12], ((uint64_t*)(m))[2]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[13], ((uint64_t*)(m))[7]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[1], ((uint64_t*)(m))[4]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[10], ((uint64_t*)(m))[5]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[10], ((uint64_t*)(m))[2]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[8], ((uint64_t*)(m))[4]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[7], ((uint64_t*)(m))[6]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[1], ((uint64_t*)(m))[5]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[15], ((uint64_t*)(m))[11]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[9], ((uint64_t*)(m))[14]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[3], ((uint64_t*)(m))[12]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[13], ((uint64_t*)(m))[0]);

    devB2B_G(v, 0, 4, 8, 12, ((uint64_t*)(m))[0], ((uint64_t*)(m))[1]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[2], ((uint64_t*)(m))[3]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[4], ((uint64_t*)(m))[5]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[6], ((uint64_t*)(m))[7]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[8], ((uint64_t*)(m))[9]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[10], ((uint64_t*)(m))[11]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t*)(m))[12], ((uint64_t*)(m))[13]);
    devB2B_G(v, 3, 4,  9, 14, ((uint64_t*)(m))[14], ((uint64_t*)(m))[15]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t*)(m))[14], ((uint64_t*)(m))[10]);
    devB2B_G(v, 1, 5, 9, 13, ((uint64_t*)(m))[4], ((uint64_t*)(m))[8]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t*)(m))[9], ((uint64_t*)(m))[15]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t*)(m))[13], ((uint64_t*)(m))[6]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t*)(m))[1], ((uint64_t*)(m))[12]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t*)(m))[0], ((uint64_t*)(m))[2]);
    devB2B_G(v, 2, 7, 8, 13, ((uint64_t*)(m))[11], ((uint64_t*)(m))[7]);
    devB2B_G(v, 3, 4, 9, 14, ((uint64_t*)(m))[5], ((uint64_t*)(m))[3]);
}


__global__ __launch_bounds__(64, 64)
__global__ void BlockMiningStep1(
    uint32_t const* data,   // Header
    uint32_t const* hashes, // precalculated hashes
    uint32_t* BHashes,      // Will be fill
    uint32_t const n_len,   // period
    uint64_t const base     // nonce base
)
{

    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t r[9] = { 0u };

    uint64_t aux[32];
    uint32_t non[algo::autolykos_v2::NONCE_SIZE_32];

    uint32_t j;
    uint64_t tmp;
    uint64_t hsh;
    uint64_t h2;
    uint32_t h3;

    for (int ii = 0; ii < 4; ii++)
    {
        tid = (algo::autolykos_v2::NONCES_PER_ITER / 4u) * ii + threadIdx.x + blockDim.x * blockIdx.x;
        if (tid < algo::autolykos_v2::NONCES_PER_ITER)
        {
            asm volatile
            (
                "add.cc.u32 %0, %1, %2;":
                "=r"(non[0]) : "r"(((uint32_t *)&base)[0]), "r"(tid)
            );

            asm volatile("addc.u32 %0, %1, 0;": "=r"(non[1]) : "r"(((uint32_t *)&base)[1]));

            ((uint32_t*)(&tmp))[0] = __byte_perm(non[1], 0, 0x0123);
            ((uint32_t*)(&tmp))[1] = __byte_perm(non[0], 0, 0x0123);

            B2B_IV(aux);
            B2B_IV(aux + 8);
            aux[0] = ivals[0];
            ((uint64_t*)(aux))[12] ^= 40;
            ((uint64_t*)(aux))[13] ^= 0;

            ((uint64_t*)(aux))[14] = ~((uint64_t*)(aux))[14];
            ((uint64_t*)(aux))[16] =  ((uint64_t*)data)[0];
            ((uint64_t*)(aux))[17] =  ((uint64_t*)data)[1];
            ((uint64_t*)(aux))[18] =  ((uint64_t*)data)[2];
            ((uint64_t*)(aux))[19] =  ((uint64_t*)data)[3];
            ((uint64_t*)(aux))[20] =  tmp;
            ((uint64_t*)(aux))[21] =  0;
            ((uint64_t*)(aux))[22] =  0;
            ((uint64_t*)(aux))[23] =  0;
            ((uint64_t*)(aux))[24] =  0;
            ((uint64_t*)(aux))[25] =  0;
            ((uint64_t*)(aux))[26] =  0;
            ((uint64_t*)(aux))[27] =  0;
            ((uint64_t*)(aux))[28] =  0;
            ((uint64_t*)(aux))[29] =  0;
            ((uint64_t*)(aux))[30] =  0;
            ((uint64_t*)(aux))[31] =  0;

            devB2B_MIX(aux, aux + 16);


            #pragma unroll
            for (j = 0; j < algo::autolykos_v2::NUM_SIZE_32; j += 2)
            {
                hsh = ivals[j >> 1];
                hsh ^= ((uint64_t*)(aux))[j >> 1] ^ ((uint64_t*)(aux))[8 + (j >> 1)];

                r[j] =  ((uint32_t*)(&hsh))[0];
                r[j + 1] = ((uint32_t*)(&hsh))[1];
            }

            //----------------------------------------------------------------------------------------
            ((uint8_t*)&h2)[0] = ((uint8_t*)r)[31];
            ((uint8_t*)&h2)[1] = ((uint8_t*)r)[30];
            ((uint8_t*)&h2)[2] = ((uint8_t*)r)[29];
            ((uint8_t*)&h2)[3] = ((uint8_t*)r)[28];
            ((uint8_t*)&h2)[4] = ((uint8_t*)r)[27];
            ((uint8_t*)&h2)[5] = ((uint8_t*)r)[26];
            ((uint8_t*)&h2)[6] = ((uint8_t*)r)[25];
            ((uint8_t*)&h2)[7] = ((uint8_t*)r)[24];

            h3 = h2 % n_len;

            #pragma unroll 8
            for (int i = 0; i < 8; ++i)
            {
                    r[7 - i] = be_u32(hashes[(h3 << 3) + i]);
            }

            //------------------------------------------------------
            B2B_IV(aux);
            B2B_IV(aux + 8);

            aux[0] = ivals[0];

            ((uint64_t*)(aux))[12] ^= 71; //31+32+8;
            ((uint64_t*)(aux))[13] ^= 0;

            ((uint64_t*)(aux))[14] = ~((uint64_t*)(aux))[14];

            uint8_t* bb = (uint8_t*)(&(((uint64_t*)(aux))[16]));
            ((uint64_t*)bb)[0] = ((uint64_t*)(&((uint8_t*)r)[1]))[0];
            ((uint64_t*)bb)[1] = ((uint64_t*)(&((uint8_t*)r)[1]))[1];
            ((uint64_t*)bb)[2] = ((uint64_t*)(&((uint8_t*)r)[1]))[2];
            ((uint64_t*)bb)[3] = ((uint64_t*)(&((uint8_t*)r)[1]))[3];

            ((uint64_t*)&bb[31])[0] = ((uint64_t*)data)[0];
            ((uint64_t*)&bb[39])[0] = ((uint64_t*)data)[1];
            ((uint64_t*)&bb[47])[0] = ((uint64_t*)data)[2];
            ((uint64_t*)&bb[55])[0] = ((uint64_t*)data)[3];

            ((uint64_t*)&bb[63])[0] = tmp;

            ((uint64_t*)(aux))[25] = 0;
            ((uint64_t*)(aux))[26] = 0;
            ((uint64_t*)(aux))[27] = 0;
            ((uint64_t*)(aux))[28] = 0;
            ((uint64_t*)(aux))[29] = 0;
            ((uint64_t*)(aux))[30] = 0;
            ((uint64_t*)(aux))[31] = 0;

            devB2B_MIX(aux, aux + 16);

            #pragma unroll
            for (j = 0; j < algo::autolykos_v2::NUM_SIZE_32; j += 2)
            {
                hsh = ivals[j >> 1];
                hsh ^= ((uint64_t*)(aux))[j >> 1] ^ ((uint64_t*)(aux))[8 + (j >> 1)];
                BHashes[algo::autolykos_v2::THREADS_PER_ITER * j + tid] = __byte_perm(((uint32_t*)(&hsh))[0], 0, 0x0123);
                BHashes[algo::autolykos_v2::THREADS_PER_ITER * (j + 1) + tid] = __byte_perm(((uint32_t*)(&hsh))[1], 0, 0x0123);
            }
        }
    }
}


__global__ __launch_bounds__(64, 64)
__global__ void BlockMiningStep2(
    t_result_64* result,    // result
    uint32_t const* data,   // Header
    uint32_t const* hashes, // precalculated hashes
    uint32_t*  BHashes,     // From BlockMiningStep1
    uint32_t const n_len,   // period
    uint32_t const height,  // block height
    uint64_t const base     // nonce
)
{
    __shared__ uint32_t shared_index[64];
    __shared__ uint32_t shared_data[512];

    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t const thread_id = threadIdx.x & 7;
    uint32_t const thrdblck_id = threadIdx.x;
    uint32_t const hash_id = threadIdx.x >> 3;

    uint64_t aux[32] = { 0ull };
    uint32_t ind[32] = { 0u };
    uint32_t r[9] = { 0u };

    uint4 v1 = { 0, 0, 0, 0 };
    uint4 v2 = { 0, 0, 0, 0 };
    uint4 v3 = { 0, 0, 0, 0 };
    uint4 v4 = { 0, 0, 0, 0 };

    uint8_t j = 0;

    if (tid < algo::autolykos_v2::NONCES_PER_ITER)
    {
        #pragma unroll
        for (int k = 0; k < 8; k++)
        {
            r[k] = (BHashes[k *  algo::autolykos_v2::THREADS_PER_ITER + tid]);
        }

        //================================================================//
        //  Generate indices
        //================================================================//
        ((uint8_t*)r)[32] = ((uint8_t*)r)[0];
        ((uint8_t*)r)[33] = ((uint8_t*)r)[1];
        ((uint8_t*)r)[34] = ((uint8_t*)r)[2];
        ((uint8_t*)r)[35] = ((uint8_t*)r)[3];

        #pragma unroll
        for (int k = 0; k < algo::autolykos_v2::K_LEN; k += 4)
        {
            ind[k]     = r[k >> 2] % n_len;
            ind[k + 1] = ((r[k >> 2] << 8) | (r[(k >> 2) + 1] >> 24)) % n_len;
            ind[k + 2] = ((r[k >> 2] << 16) | (r[(k >> 2) + 1] >> 16)) % n_len;
            ind[k + 3] = ((r[k >> 2] << 24) | (r[(k >> 2) + 1] >> 8)) % n_len;
        }

        //================================================================//
        //  Calculate result
        //================================================================//
        shared_index[thrdblck_id] = ind[0];
        __syncthreads();

        shared_data[(hash_id << 3) + thread_id]       = (hashes[(shared_index[hash_id] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 64]  = (hashes[(shared_index[hash_id + 8] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 128] = (hashes[(shared_index[hash_id + 16] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 192] = (hashes[(shared_index[hash_id + 24] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 256] = (hashes[(shared_index[hash_id + 32] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 320] = (hashes[(shared_index[hash_id + 40] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 384] = (hashes[(shared_index[hash_id + 48] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 448] = (hashes[(shared_index[hash_id + 56] << 3) + thread_id]);
        __syncthreads();

        v1.x = shared_data[(thrdblck_id << 3) + 0];
        v1.y = shared_data[(thrdblck_id << 3) + 1];
        v1.z = shared_data[(thrdblck_id << 3) + 2];
        v1.w = shared_data[(thrdblck_id << 3) + 3];
        v3.x = shared_data[(thrdblck_id << 3) + 4];
        v3.y = shared_data[(thrdblck_id << 3) + 5];
        v3.z = shared_data[(thrdblck_id << 3) + 6];
        v3.w = shared_data[(thrdblck_id << 3) + 7];

        shared_index[thrdblck_id] = ind[1];
        __syncthreads();

        shared_data[(hash_id << 3) + thread_id]       = (hashes[(shared_index[hash_id] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 64]  = (hashes[(shared_index[hash_id + 8] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 128] = (hashes[(shared_index[hash_id + 16] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 192] = (hashes[(shared_index[hash_id + 24] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 256] = (hashes[(shared_index[hash_id + 32] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 320] = (hashes[(shared_index[hash_id + 40] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 384] = (hashes[(shared_index[hash_id + 48] << 3) + thread_id]);
        shared_data[(hash_id << 3) + thread_id + 448] = (hashes[(shared_index[hash_id + 56] << 3) + thread_id]);
        __syncthreads();

        v2.x = shared_data[(thrdblck_id << 3) + 0];
        v2.y = shared_data[(thrdblck_id << 3) + 1];
        v2.z = shared_data[(thrdblck_id << 3) + 2];
        v2.w = shared_data[(thrdblck_id << 3) + 3];
        v4.x = shared_data[(thrdblck_id << 3) + 4];
        v4.y = shared_data[(thrdblck_id << 3) + 5];
        v4.z = shared_data[(thrdblck_id << 3) + 6];
        v4.w = shared_data[(thrdblck_id << 3) + 7];

        asm volatile ("add.cc.u32 %0, %1, %2;":"=r"(r[0]) : "r"(v1.x), "r"(v2.x));
        asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[1]) : "r"(v1.y), "r"(v2.y));
        asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[2]) : "r"(v1.z), "r"(v2.z));
        asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[3]) : "r"(v1.w), "r"(v2.w));
        asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[4]) : "r"(v3.x), "r"(v4.x));
        asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[5]) : "r"(v3.y), "r"(v4.y));
        asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[6]) : "r"(v3.z), "r"(v4.z));
        asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[7]) : "r"(v3.w), "r"(v4.w));
        asm volatile ("addc.u32 %0, 0, 0;": "=r"(r[8]));

        //////////////////////////////////////////////////////////////////////////////////////////////////////////
        // remaining additions
        #pragma unroll
        for (int k = 2; k < algo::autolykos_v2::K_LEN; ++k)
        {
            shared_index[thrdblck_id] = ind[k];
            __syncthreads();

            shared_data[(hash_id << 3) + thread_id]       = (hashes[(shared_index[hash_id] << 3) + thread_id]);
            shared_data[(hash_id << 3) + thread_id + 64]  = (hashes[(shared_index[hash_id + 8] << 3) + thread_id]);
            shared_data[(hash_id << 3) + thread_id + 128] = (hashes[(shared_index[hash_id + 16] << 3) + thread_id]);
            shared_data[(hash_id << 3) + thread_id + 192] = (hashes[(shared_index[hash_id + 24] << 3) + thread_id]);
            shared_data[(hash_id << 3) + thread_id + 256] = (hashes[(shared_index[hash_id + 32] << 3) + thread_id]);
            shared_data[(hash_id << 3) + thread_id + 320] = (hashes[(shared_index[hash_id + 40] << 3) + thread_id]);
            shared_data[(hash_id << 3) + thread_id + 384] = (hashes[(shared_index[hash_id + 48] << 3) + thread_id]);
            shared_data[(hash_id << 3) + thread_id + 448] = (hashes[(shared_index[hash_id + 56] << 3) + thread_id]);
            __syncthreads();

            v1.x = shared_data[(thrdblck_id << 3) + 0];
            v1.y = shared_data[(thrdblck_id << 3) + 1];
            v1.z = shared_data[(thrdblck_id << 3) + 2];
            v1.w = shared_data[(thrdblck_id << 3) + 3];
            v2.x = shared_data[(thrdblck_id << 3) + 4];
            v2.y = shared_data[(thrdblck_id << 3) + 5];
            v2.z = shared_data[(thrdblck_id << 3) + 6];
            v2.w = shared_data[(thrdblck_id << 3) + 7];

            asm volatile ("add.cc.u32 %0, %0, %1;":"+r"(r[0]) : "r"(v1.x));
            asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[1]) : "r"(v1.y));
            asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[2]) : "r"(v1.z));
            asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[3]) : "r"(v1.w));
            asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[4]) : "r"(v2.x));
            asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[5]) : "r"(v2.y));
            asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[6]) : "r"(v2.z));
            asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[7]) : "r"(v2.w));
            asm volatile ("addc.u32 %0, %0, 0;": "+r"(r[8]));
        }

        B2B_IV(aux);
        B2B_IV(aux + 8);
        aux[0] = ivals[0];
        ((uint64_t*)(aux))[12] ^= 32;
        ((uint64_t*)(aux))[13] ^= 0;

        ((uint64_t*)(aux))[14] = ~((uint64_t*)(aux))[14];

        uint8_t* bb = (uint8_t*)(&(((uint64_t*)(aux))[16]));
        for (j = 0; j < algo::autolykos_v2::NUM_SIZE_8; ++j)
        {
            bb[j] = ((const uint8_t*)r)[algo::autolykos_v2::NUM_SIZE_8 - j - 1];
        }

        ((uint64_t*)(aux))[20] = 0;
        ((uint64_t*)(aux))[21] = 0;
        ((uint64_t*)(aux))[22] = 0;
        ((uint64_t*)(aux))[23] = 0;
        ((uint64_t*)(aux))[24] = 0;
        ((uint64_t*)(aux))[25] = 0;
        ((uint64_t*)(aux))[26] = 0;
        ((uint64_t*)(aux))[27] = 0;
        ((uint64_t*)(aux))[28] = 0;
        ((uint64_t*)(aux))[29] = 0;
        ((uint64_t*)(aux))[30] = 0;
        ((uint64_t*)(aux))[31] = 0;

        devB2B_MIX(aux, aux + 16);

        uint64_t hsh;
        uint32_t r_l[32];
        #pragma unroll
        for (j = 0; j < algo::autolykos_v2::NUM_SIZE_32; j += 2)
        {
            hsh = ivals[j >> 1];
            hsh ^= ((uint64_t*)(aux))[j >> 1] ^ ((uint64_t*)(aux))[8 + (j >> 1)];
            r_l[j] = ((uint32_t*)&hsh)[0];
            r_l[j+1] = ((uint32_t*)&hsh)[1];

        }

        #pragma unroll 32
        for (j = 0; j < algo::autolykos_v2::NUM_SIZE_8; j ++)
        {
            ((uint8_t*)r)[j] = ((uint8_t*)r_l)[algo::autolykos_v2::NUM_SIZE_8 - j - 1];
        }


        //================================================================//
        //  Dump result to global memory -- LITTLE ENDIAN
        //================================================================//
        j =    ((uint64_t*)r)[3] < ((uint64_t*)bound_)[3]
            || ((uint64_t*)r)[3] == ((uint64_t*)bound_)[3]
            &&
            (
                   ((uint64_t*)r)[2] < ((uint64_t*)bound_)[2]
                || ((uint64_t*)r)[2] == ((uint64_t*)bound_)[2]
                &&
                (
                       ((uint64_t*)r)[1] < ((uint64_t*)bound_)[1]
                    || ((uint64_t*)r)[1] == ((uint64_t*)bound_)[1]
                    && ((uint64_t*)r)[0] < ((uint64_t*)bound_)[0]
                )
            );

        if (j)
        {
            uint32_t const index = atomicAdd((uint32_t*)&result->index, 1);
            if (index < MAX_RESULT_INDEX)
            {
                result->found = true;
                result->nonce[index] = tid + base;
            }
        }
    }
}


__host__
bool autolykos_v2_mhssamadi_init(algo::hash256 const& boundary)
{
    CUDA_ER(hipMemcpyToSymbol(HIP_SYMBOL(bound_), (void*)&boundary, algo::LEN_HASH_256));

    return true;
}


__host__
bool autolykos_v2_mhssamadi(
        hipStream_t stream,
        t_result_64* result,
        uint32_t const* dag,
        uint32_t* BHashes,
        uint32_t* header,
        uint32_t const blocks,
        uint32_t const threads,
        uint32_t const period,
        uint32_t const height)
{
    uint64_t const nonce{ 11055774138563218679ull };

    BlockMiningStep1<<<blocks / 4u, threads, 0, stream>>>
    (
        header,
        dag,
        BHashes,
        period,
        nonce
    );
    CUDA_ER(hipStreamSynchronize(stream));
    CUDA_ER(hipGetLastError());

    BlockMiningStep2<<<blocks, threads, 0, stream>>>
    (
        result,
        header,
        dag,
        BHashes,
        period,
        height,
        nonce
    );
    CUDA_ER(hipStreamSynchronize(stream));
    CUDA_ER(hipGetLastError());

    return true;
}
